
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
      fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
        msg, hipGetErrorString(__err), \
        __FILE__, __LINE__); \
      fprintf(stderr, "*** FAILED - ABORTING\n"); \
      exit(1); \
    } \
  } while (0)

const int DSIZE = 8192;
const int block_size = 32; // The CUDA max is 1024 threads per block
const float A_val = 3.0f;
const float B_val = 2.0f;

__global__ void mmul(const float *A, const float *B, float *C, int ds) {
  __shared__ float As[block_size][block_size];
  __shared__ float Bs[block_size][block_size];

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if ((idx < ds) && (idy < ds)) {
    float temp = 0;
    for (int i = 0; i < ds/block_size; i++) {
      //As[threadIdx.y][threadIdx.x] = A[idy*idx+i];
      //Bs[threadIdx.y][threadIdx.x] = B[idy*idx+i];
      As[threadIdx.y][threadIdx.x] = A[idy*ds + (i*block_size + threadIdx.x)];
      Bs[threadIdx.y][threadIdx.x] = B[(i*block_size + threadIdx.y) * ds + idx];
      __syncthreads();

      for (int k = 0; k < block_size; k++) {
        //temp += As[idy][k+idx] * Bs[k+idy][idx];
	temp += As[threadIdx.y][k] * Bs[k][threadIdx.x];
      }
      __syncthreads();
    }
    C[idy*ds+idx] = temp;
  }
}

int main() {
  float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
  clock_t t0, t1, t2;
  double t1sum = 0.0;
  double t2sum = 0.0;

  t0 = clock();

  h_A = new float[DSIZE*DSIZE];
  h_B = new float[DSIZE*DSIZE];
  h_C = new float[DSIZE*DSIZE];
  for (int i = 0; i < DSIZE*DSIZE; i++) {
    h_A[i] = A_val;
    h_B[i] = B_val;
    h_C[i] = 0.0;
  }

  t1 = clock();
  t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
  printf("Init took %f seconds. Begin compute.\n", t1sum);

  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failture");

  dim3 block(block_size, block_size);
  dim3 grid((DSIZE+block.x-1)/block.x, (DSIZE+block.y-1)/block.y);
  mmul<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
  cudaCheckErrors("kernel launch failure");

  hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);

  t2 = clock();
  t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
  printf("Done. Compute took %f seconds\n", t2sum);

  cudaCheckErrors("Kernel execution failure or hipMemcpy H2D failure");
  for (int i = 0; i < DSIZE*DSIZE; i++) { 
    if (h_C[i] != A_val*B_val*DSIZE) {
      printf("mismatch at index %d, was: %f, should be: %f\n", 
		      i, h_C[i], A_val*B_val*DSIZE); 
      return -1;
    }
  }
  printf("Success!\n");
  return 0;
}
